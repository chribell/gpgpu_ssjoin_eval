#include "hip/hip_runtime.h"
/*********************************************************************
11
12	 Copyright (C) 2015 by Wisllay Vitrio
42	 Copyright (C) 2017 by Sidney Ribeiro Junior
13
14	 This program is free software; you can redistribute it and/or modify
15	 it under the terms of the GNU General Public License as published by
16	 the Free Software Foundation; either version 2 of the License, or
17	 (at your option) any later version.
18
19	 This program is distributed in the hope that it will be useful,
20	 but WITHOUT ANY WARRANTY; without even the implied warranty of
21	 MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
22	 GNU General Public License for more details.
23
24	 You should have received a copy of the GNU General Public License
25	 along with this program; if not, write to the Free Software
26	 Foundation, Inc., 675 Mass Ave, Cambridge, MA 02139, USA.
27
28	 ********************************************************************/

#include <vector>
#include <string>
#include <sstream>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#ifdef __linux
#include <sys/time.h>

#else 
#include <sys/timeb.h>
#include <time.h>

#include <windows.h>


#endif

#ifdef __linux

double gettime() {
	timespec ts;
	clock_gettime(CLOCK_REALTIME, &ts);
	return double(ts.tv_sec) + double(ts.tv_nsec) / 1e9;
}
#else
#include <windows.h>
double gettime() { // granularity about 50 microsecs on my machine
	static LARGE_INTEGER freq, start;
	LARGE_INTEGER count;
	if (!QueryPerformanceCounter(&count))
		// FatalError("QueryPerformanceCounter");
		fprintf(stderr, "QueryPerformanceCounter");
	if (!freq.QuadPart) { // one time initialization
		if (!QueryPerformanceFrequency(&freq))
			//FatalError("QueryPerformanceFrequency");
			fprintf(stderr, "QueryPerformanceCounter");
		start = count;
	}
	return (double)(count.QuadPart - start.QuadPart) / freq.QuadPart;
}
#endif

#include "utils.cuh"

int WARP_SIZE = 32;

vector<string> &split(const string &s, char delim, vector<string> &elems) {
	stringstream ss(s);
	string item;
	while (getline(ss, item, delim)) {
		elems.push_back(item);
	}
	return elems;
}

vector<string> split(const string &s, char delim) {
	vector<string> elems;
	split(s, delim, elems);
	return elems;
}

void get_grid_config(dim3 &grid, dim3 &threads) {
	//get the device properties
	static bool flag = 0;
	static dim3 lgrid, lthreads;
	if (!flag){
		hipDeviceProp_t devProp;
		hipGetDeviceProperties(&devProp, omp_get_thread_num() / NUM_STREAMS);

		//Adjust the grid dimensions based on the device properties
		int num_blocks = devProp.multiProcessorCount;
		lgrid = dim3(num_blocks*16);
		lthreads = dim3(devProp.maxThreadsPerBlock/2);
		//lgrid = dim3(8);
		//lthreads = dim3(512);
		flag = 1;
	}
	grid = lgrid;
	threads = lthreads;
}

void __gpuAssert(hipError_t stat, int line, string file) {
	if (stat != hipSuccess) {
		fprintf(stderr, "Error %s at line %d in file %s\n",
			hipGetErrorString(stat), line, file.c_str());
		exit(1);
	}
}

float get_minoverlap_jaccard(int size_x1, int size_x2, double threshold) {
	return (threshold*((float) size_x1 + size_x2)) / (1.0 + threshold);
}

float get_minsize_jaccard(int size, double threshold) {
	return threshold * ((float) size);
}

int get_midprefix(int size, double threshold) {
	return size - ceil(get_minoverlap_jaccard(size, size, threshold)) + 1;
}

int get_maxprefix(int size, double threshold) {
	return size - ceil(get_minsize_jaccard(size, threshold)) + 1;
}

int max_size_jaccard(int size, double threshold) {
	return ceil(((float) size)/threshold);
}
