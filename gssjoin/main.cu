#include "hip/hip_runtime.h"
/*********************************************************************
11
12	 Copyright (C) 2016 by Sidney Ribeiro Junior
13
14	 This program is free software; you can redistribute it and/or modify
15	 it under the terms of the GNU General Public License as published by
16	 the Free Software Foundation; either version 2 of the License, or
17	 (at your option) any later version.
18
19	 This program is distributed in the hope that it will be useful,
20	 but WITHOUT ANY WARRANTY; without even the implied warranty of
21	 MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
22	 GNU General Public License for more details.
23
24	 You should have received a copy of the GNU General Public License
25	 along with this program; if not, write to the Free Software
26	 Foundation, Inc., 675 Mass Ave, Cambridge, MA 02139, USA.
27
28	 ********************************************************************/

#define CUDA_API_PER_THREAD_DEFAULT_STREAM

#include "gpu.h"

#include <vector>
#include <fstream>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <algorithm>
#include <iostream>
#include <omp.h>
#include <string>
#include <sstream>
#include <hip/hip_runtime.h>
#include <map>

#include "structs.cuh"
#include "utils.cuh"
#include "inverted_index.cuh"
#include "simjoin.cuh"
#include "device_timing.hxx"

#define NUM_STREAMS 1

using namespace std;

struct FileStats {
	int num_sets;
	int num_terms;

	vector<int> sizes; // set sizes
	vector<int> start; // beginning of each entrie

	FileStats() : num_sets(0), num_terms(0) {}
};

FileStats readInputFile(string &file, vector<Entry> &entries);
void processTestFile(InvertedIndex &index, FileStats &stats, string &file, float threshold, stringstream &fileout, bool aggregate, DeviceTiming& deviceTiming);


int biggestQuerySize = -1;

int gpu(int argc, char **argv) {

    if (argc != 6) {
        cerr
                << "Wrong parameters. Correct usage: <executable> <input_file> <threshold> <output_file> <number_of_gpus> <aggregate>"
                << endl;
        exit(1);
    }

    int gpuNum;
    hipGetDeviceCount(&gpuNum);

    if (gpuNum > atoi(argv[4])) {
        gpuNum = atoi(argv[4]);
        if (gpuNum < 1)
            gpuNum = 1;
    }
    bool aggregate = atoi(argv[5]) == 1;
    //cerr << "Using " << gpuNum << "GPUs" << endl;

    // we use 2 streams per GPU
    int numThreads = gpuNum * NUM_STREAMS;

    omp_set_num_threads(numThreads);

    //truncate output files
    ofstream ofsf(argv[3], ofstream::trunc);
    ofsf.close();

    ofstream ofsfileoutput(argv[3], ofstream::out | ofstream::app);

	vector<string> inputs;// to read the whole test file in memory
	vector<InvertedIndex> indexes;
	indexes.resize(gpuNum);

	double starts, ends;

	string inputFileName(argv[1]);

	printf("Reading file...\n");
	vector<Entry> entries;

	starts = gettime();
	FileStats stats = readInputFile(inputFileName, entries);
	ends = gettime();

	printf("Time taken: %lf seconds\n", ends - starts);
	DeviceTiming deviceTiming;
	vector<stringstream*> outputString;
	//Each thread builds an output string, so it can be flushed at once at the end of the program
	for (int i = 0; i < numThreads; i++){
		outputString.push_back(new stringstream);
	}

	//create an inverted index for all streams in each GPU
	#pragma omp parallel num_threads(gpuNum)
	{
		int cpuid = omp_get_thread_num();
		hipSetDevice(cpuid);
		double start, end;

		start = gettime();
		indexes[cpuid] = make_inverted_index(stats.num_sets, stats.num_terms, entries);
		end = gettime();

		#pragma omp single nowait
		printf("Total time taken for insertion: %lf seconds\n", end - start);
	}


	#pragma omp parallel 
	{
		int cpuid = omp_get_thread_num();
		hipSetDevice(cpuid / NUM_STREAMS);

		float threshold = atof(argv[2]);

		FileStats lstats = stats;

		processTestFile(indexes[cpuid / NUM_STREAMS], lstats, inputFileName, threshold, *outputString[cpuid], aggregate, deviceTiming);
		if (cpuid %  NUM_STREAMS == 0)
			gpuAssert(hipDeviceReset());

	}

    if (!aggregate) {
        starts = gettime();
        for (int i = 0; i < numThreads; i++){
            ofsfileoutput << outputString[i]->str();
        }
        ends = gettime();
        printf("Time taken to write output: %lf seconds\n", ends - starts);
    }

    ofsfileoutput.close();

    return 0;
}

FileStats readInputFile(string &filename, vector<Entry> &entries) {
	ifstream input(filename.c_str());
	string line;

	FileStats stats;
	int accumulatedsize = 0;
	int set_id = 0;

	while (!input.eof()) {
		getline(input, line);
		if (line == "") continue;

		vector<string> tokens = split(line, ' ');
		biggestQuerySize = max((int)tokens.size(), biggestQuerySize);

		int size = tokens.size();
		stats.sizes.push_back(size);
		stats.start.push_back(accumulatedsize);
		accumulatedsize += size;

		for (int i = 0; i < size; i++) {
			int term_id = atoi(tokens[i].c_str());
			stats.num_terms = max(stats.num_terms, term_id + 1);
			entries.push_back(Entry(set_id, term_id));
		}
		set_id++;
	}

	stats.num_sets = stats.start.size();

	input.close();

	return stats;
}

void allocVariables(DeviceVariables *dev_vars, float threshold, int num_sets, Similarity** distances){
	dim3 grid, threads;

	get_grid_config(grid, threads);

	gpuAssert(hipMalloc(&dev_vars->d_dist, num_sets * sizeof(Similarity))); // distance between all the sets and the query doc
	gpuAssert(hipMalloc(&dev_vars->d_result, num_sets * sizeof(Similarity))); // compacted similarities between all the sets and the query doc
	gpuAssert(hipMalloc(&dev_vars->d_sim, num_sets * sizeof(int))); // count of elements in common
	gpuAssert(hipMalloc(&dev_vars->d_sizes, num_sets * sizeof(int))); // size of all sets
	gpuAssert(hipMalloc(&dev_vars->d_query, biggestQuerySize * sizeof(Entry))); // query
	gpuAssert(hipMalloc(&dev_vars->d_index, biggestQuerySize * sizeof(int)));
	gpuAssert(hipMalloc(&dev_vars->d_count, biggestQuerySize * sizeof(int)));

	*distances = (Similarity*)malloc(num_sets * sizeof(Similarity));

	int blocksize = 1024;
	int numBlocks = num_sets / blocksize + (num_sets % blocksize ? 1 : 0);

	gpuAssert(hipMalloc(&dev_vars->d_bC,sizeof(int)*(numBlocks + 1)));
	gpuAssert(hipMalloc(&dev_vars->d_bO,sizeof(int)*numBlocks));

}

void freeVariables(DeviceVariables *dev_vars, InvertedIndex &index, Similarity** distances){
	hipFree(dev_vars->d_dist);
	hipFree(dev_vars->d_result);
	hipFree(dev_vars->d_sim);
	hipFree(dev_vars->d_sizes);
	hipFree(dev_vars->d_query);
	hipFree(dev_vars->d_index);
	hipFree(dev_vars->d_count);
	hipFree(dev_vars->d_bC);
	hipFree(dev_vars->d_bO);

	free(*distances);

	if (omp_get_thread_num() % NUM_STREAMS == 0){
		hipFree(index.d_count);
		hipFree(index.d_index);
		hipFree(index.d_inverted_index);
	}
}

void processTestFile(InvertedIndex &index, FileStats &stats, string &filename, float threshold, stringstream &outputfile, bool aggregate, DeviceTiming& deviceTiming) {

	int num_test_local = 0, setid;

	//#pragma omp single nowait
	size_t finalResult = 0;
	DeviceVariables dev_vars;
	Similarity* distances;

	allocVariables(&dev_vars, threshold, index.num_sets, &distances);

	hipMemcpyAsync(dev_vars.d_sizes, &stats.sizes[0], index.num_sets * sizeof(int), hipMemcpyHostToDevice);

	double start = gettime();

#pragma omp for
	for (setid = 0; setid < index.num_sets - 1; setid++){

		num_test_local++;

		int totalSimilars = findSimilars(index, threshold, &dev_vars, distances, setid, stats.start[setid], stats.sizes[setid], aggregate, deviceTiming);
        finalResult += totalSimilars;
        if (!aggregate) {
            for (int i = 0; i < totalSimilars; i++) {
                outputfile << "(" << setid << ", " << distances[i].set_id << "): "
                    << distances[i].similarity << endl;
            }
        }
    }

	freeVariables(&dev_vars, index, &distances);
	int threadid = omp_get_thread_num();

	#pragma omp barrier

	double end = gettime();

    std::cout
        << "Result: " << finalResult << std::endl
        << "Runtime: " << end - start << " secs" << std::endl;

    std::cout << deviceTiming;
}
