#include "hip/hip_runtime.h"
/*********************************************************************
11
12	 Copyright (C) 2015 by Wisllay Vitrio
13
14	 This program is free software; you can redistribute it and/or modify
15	 it under the terms of the GNU General Public License as published by
16	 the Free Software Foundation; either version 2 of the License, or
17	 (at your option) any later version.
18
19	 This program is distributed in the hope that it will be useful,
20	 but WITHOUT ANY WARRANTY; without even the implied warranty of
21	 MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
22	 GNU General Public License for more details.
23
24	 You should have received a copy of the GNU General Public License
25	 along with this program; if not, write to the Free Software
26	 Foundation, Inc., 675 Mass Ave, Cambridge, MA 02139, USA.
27
28	 ********************************************************************/

/* *
 * knn.cu
 */

#define CUDA_API_PER_THREAD_DEFAULT_STREAM

#include <stdio.h>
#include <stdlib.h>
#include <algorithm>
#include <iostream>
#include <queue>
#include <vector>
#include <set>
#include <functional>

#include "simjoin.cuh"
#include "structs.cuh"
#include "utils.cuh"
#include "inverted_index.cuh"
#include "cuCompactor.cuh"

struct is_bigger_than_threshold
{
	float threshold;
	is_bigger_than_threshold(float thr) : threshold(thr) {};
	__host__ __device__
	bool operator()(const Similarity &reg)
	{
		return (reg.similarity > threshold);
	}
};


__host__ int findSimilars(InvertedIndex inverted_index, float threshold, struct DeviceVariables *dev_vars, Similarity* distances,
		int docid, int querystart, int querysize, bool aggregate, DeviceTiming& deviceTiming) {

	dim3 grid, threads;
	get_grid_config(grid, threads);

	int num_sets = inverted_index.num_sets - docid - 1;
	int *d_count = dev_vars->d_count, *d_index = dev_vars->d_index, *d_sim = dev_vars->d_sim, *size_doc = dev_vars->d_sizes;
	int *d_BlocksCount = dev_vars->d_bC, *d_BlocksOffset = dev_vars->d_bO;
	Entry *d_query = inverted_index.d_entries + querystart;
	Similarity *d_similarity = dev_vars->d_dist, *d_result = dev_vars->d_result;

    DeviceTiming::EventPair* memSet = deviceTiming.add("Mem set", 0);
	gpuAssert(hipMemset(d_sim + docid + 1, 0, num_sets*sizeof(int)));
    deviceTiming.finish(memSet);

    DeviceTiming::EventPair* termCount = deviceTiming.add("Term count", 0);
	get_term_count_and_tf_idf<<<grid, threads>>>(inverted_index, d_query, d_count, querysize);
    deviceTiming.finish(termCount);

	thrust::device_ptr<int> thrust_d_count(d_count);
	thrust::device_ptr<int> thrust_d_index(d_index);
	thrust::inclusive_scan(thrust_d_count, thrust_d_count + querysize, thrust_d_index);

    DeviceTiming::EventPair* calcJacc = deviceTiming.add("Calculate Jaccard", 0);
	calculateJaccardSimilarity<<<grid, threads>>>(inverted_index, d_query, d_index, d_sim, querysize, docid);
    deviceTiming.finish(calcJacc);

    DeviceTiming::EventPair* filterReg = deviceTiming.add("Filter registers", 0);
	filter_registers<<<grid, threads>>>(d_sim, threshold, querysize, docid, inverted_index.num_sets, size_doc, d_similarity);
    deviceTiming.finish(filterReg);

	int blocksize = 1024;
	int numBlocks = cuCompactor::divup(num_sets, blocksize);

    DeviceTiming::EventPair* compactSimilars = deviceTiming.add("Compact similars", 0);
	int totalSimilars = cuCompactor::compact2<Similarity>(d_similarity + docid + 1, d_result, num_sets, is_bigger_than_threshold(threshold), blocksize, numBlocks, d_BlocksCount, d_BlocksOffset);
    deviceTiming.finish(compactSimilars);

    DeviceTiming::EventPair* transferPairs = deviceTiming.add("Transfer pairs", 0);
	if (totalSimilars && !aggregate) hipMemcpyAsync(distances, d_result, sizeof(Similarity)*totalSimilars, hipMemcpyDeviceToHost);
    deviceTiming.finish(transferPairs);

	return totalSimilars;
}

__global__ void calculateJaccardSimilarity(InvertedIndex inverted_index, Entry *d_query, int *index, int *dist, int D, int docid) {
	__shared__ int N;

	if (threadIdx.x == 0) {
		N = index[D - 1];	//Total number of items to be queried
	}
	__syncthreads();

	int block_size = N / gridDim.x + (N % gridDim.x == 0 ? 0 : 1);		//Partition size
	int lo = block_size * (blockIdx.x); 								//Beginning of the block
	int hi = min(lo + block_size, N); 								//End of the block
	int size = hi - lo;											// Real partition size (the last one can be smaller)

	int idx = 0;
	int end;

	for (int i = threadIdx.x; i < size; i += blockDim.x) {
		int pos = i + lo;

		while (true) {
			end = index[idx];

			if (end <= pos) {
				idx++;
			}
			else {
				break;
			}
		}

		Entry entry = d_query[idx]; 		//finds out the term
		int offset = end - pos;

		int idx2 = inverted_index.d_index[entry.term_id] - offset;
		Entry index_entry = inverted_index.d_inverted_index[idx2];

		if (index_entry.set_id > docid) {
			atomicAdd(&dist[index_entry.set_id], 1);
		}
	}
}


__global__ void get_term_count_and_tf_idf(InvertedIndex inverted_index, Entry *query, int *count, int N) {
	int block_size = N / gridDim.x + (N % gridDim.x == 0 ? 0 : 1);		//Partition size
	int offset = block_size * (blockIdx.x); 				//Beginning of the block
	int lim = min(offset + block_size, N); 					//End of the block
	int size = lim - offset; 						//Block size

	query += offset;
	count += offset;

	for (int i = threadIdx.x; i < size; i += blockDim.x) {
		Entry entry = query[i];

		int idf = inverted_index.d_count[entry.term_id];
		//query[i].tf_idf = entry.tf * log(inverted_index.num_sets / float(max(1, idf)));
		count[i] = idf;
		//atomicAdd(d_qnorm, query[i].tf_idf * query[i].tf_idf);
		//atomicAdd(d_qnorml1, query[i].tf_idf);
	}
}

__global__ void filter_registers(int *sim, float threshold, int querysize, int docid, int N, int *doc_size, Similarity *similars) { // similars + id_doc
	N -= (docid + 1);
	int block_size = N / gridDim.x + (N % gridDim.x == 0 ? 0 : 1);		//Partition size
	int offset = block_size * (blockIdx.x) + docid + 1; 				//Beginning of the block
	int lim = min(offset + block_size, N + docid + 1); 					//End of the block
	int size = lim - offset;

	similars += offset;
	sim += offset;
	doc_size += offset;

	for (int i = threadIdx.x; i < size; i += blockDim.x) {
		float jac = sim[i]/ (float) (querysize + doc_size[i] - sim[i]);

		similars[i].set_id = offset + i;
		similars[i].similarity = jac;
	}
}
